#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <coding.cuh>
#include <cuda_ext.cuh>
#include <gpu.cuh>
#include <log.hpp>
#include <math_ext.hpp>
#include <operators.cuh>
#include <signal.hpp>

#include "gpu_fx.cu"

__device__ inline hipfftComplex conjugate(hipfftComplex v) { return {v.x, -v.y}; }
__device__ inline hipfftComplex timesj(hipfftComplex v) { return {-v.y, v.x}; }

__global__ static void ccc_unpackCto2C(hipfftComplex* dest1, hipfftComplex* dest2, const hipfftComplex* src, size_t fftSize) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < fftSize / 2; s += stride.x) {
        auto idxa = s;
        auto idxb = (fftSize - s);

        auto va = src[idxa];
        auto vb = s ? conjugate(src[idxb]) : va;
        auto la = 0.5f * (va + vb);
        auto lb = timesj(-0.5f * (va - vb));

        dest1[idxa] = la;
        dest2[idxa] = lb;
        if (s) {
            dest1[idxb] = conjugate(la);
            dest2[idxb] = conjugate(lb);
        }
    }
}

__global__ static void ccc_complexMultiplyAndScale(hipfftComplex* dst, const hipfftComplex* src1, const hipfftComplex* src2, size_t n, float scale) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < n; s += stride.x) {
        dst[s] = hipCmulf(src1[s], src2[s]) * scale;
    }
}

__global__ static void f2f2f2_multiplyAndScale(float2* dst, const float2* src1, const float2* src2, size_t n, float scale) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < n; s += stride.x) {
        dst[s] = src1[s] * src2[s] * scale;
    }
}

__global__ static void f2f2f2_pointwiseAdd(float2* dst, float2* src1, const float2* src2, size_t n) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < n; s += stride.x) {
        dst[s] = src1[s] + src2[s];
    }
}

__global__ static void f2fff2_pointwiseAdd(float2* dst, float* src1x, float* src1y, const float2* src2, size_t n) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < n; s += stride.x) {
        dst[s].x = src1x[s] + src2[s].x;
        dst[s].y = src1y[s] + src2[s].y;
    }
}

__global__ static void fff_pointwiseAdd(float* dst, float* src1, const float* src2, size_t n) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < n; s += stride.x) {
        dst[s] = src1[s] + src2[s];
    }
}

__global__ static void f2f2f2_mix(float2* dst, const float2* src1, const float2* src2, size_t n, float ratio) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < n; s += stride.x) {
        dst[s].x = src1[s].x * (1 - ratio) + src2[s].x * ratio;
        dst[s].y = src1[s].y * (1 - ratio) + src2[s].y * ratio;
    }
}

__global__ static void fff_mix(float* dst, const float* src1, const float* src2, size_t n, float ratio) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < n; s += stride.x) {
        dst[s] = src1[s] * (1 - ratio) + src2[s] * ratio;
    }
}

__global__ static void f2_scale(float2* data, size_t n, float scale) {
    auto stride = gridDim * blockDim;
    auto offset = blockDim * blockIdx + threadIdx;

    for (auto s = offset.x; s < n; s += stride.x) {
        data[s] *= scale;
    }
}

class FxConvFd : public GpuFx {
   protected:
    IPCMSignal* _ir_signal;

    dim3 _nBlocks;
    dim3 _nThreads;
    size_t _nSharedMem;

    size_t _fft_size;
    size_t _fft_size_non_redundant;
    int _ir_db_scale;
    float _mix_ratio;
    bool _force_wet_mix;

    IMemCpyNode* _src_node = nullptr;
    IKernelNode* _dest_node = nullptr;
    char* _ir_byte_buf = nullptr;

    float getIRAttenuationFactor() const {
        return pow(10, _ir_db_scale * 0.05);  // IRs are usually too loud, so we scale them by _ir_db_scale dB
    }

    size_t getMinIROrFFTSize() const {
        return std::min(_ir_signal->getFrameCount(), _fft_size);
    }

    virtual void allocateBuffers() = 0;
    virtual void deallocateBuffers() = 0;
    virtual hipStream_t _process(hipStream_t stream, float* src, const float* dst, hipStreamCaptureStatus capture_status) = 0;

   public:
    FxConvFd(std::string name, IPCMSignal* ir_signal, size_t max_ir_size, int ir_db_scale, bool force_wet_mix) : GpuFx(name), _nBlocks(256), _nThreads(256), _nSharedMem(0), _ir_signal(ir_signal), _ir_db_scale(ir_db_scale), _mix_ratio(0.5), _force_wet_mix(force_wet_mix) {
        if (max_ir_size == 0) {
            max_ir_size = _ir_signal->getFrameCount();
        }
        _fft_size = roundUpToPow2(std::min(_ir_signal->getFrameCount(), max_ir_size));
        _fft_size_non_redundant = _fft_size / 2 + 1;
        if (_ir_signal->getFrameCount() > _fft_size) {
            spdlog::warn("IR file given to {} is longer than specified max fft size of {}. Truncating to {} samples.", name, max_ir_size, _fft_size);
        }
    }

    virtual ~FxConvFd() {
        delete _ir_signal;
    }

    void configure(size_t process_buffer_size, size_t n_input_channels, size_t n_output_channels) override {
        if (n_input_channels != 0 && n_input_channels != _n_in_channels || n_output_channels != 0 && n_output_channels != _n_out_channels) {
            spdlog::warn("{} is a fixed channel fx ({}i{}). The configured channel count will be ignored {}i{}.", _name, _n_in_channels, _n_out_channels, n_input_channels, n_output_channels);
        }
        GpuFx::configure(process_buffer_size, _n_in_channels, _n_out_channels);
    }

    void updateBufferPtrs(hipGraphExec_t procGraphExec, const BufferRack* dst, const BufferRack* src) override {
        _src_node->updateSrcPtr(src->getDataMod(), procGraphExec);
        _dest_node->updateKernelParamAt(0, dst->getDataMod(), procGraphExec);
    }

    hipStream_t process(hipStream_t stream, const BufferRack* dst, const BufferRack* src, hipStreamCaptureStatus capture_status) override {
        return _process(stream, dst->getDataMod(), src->getDataMod(), capture_status);
    }

    virtual void teardown() override {
        if (_src_node) delete _src_node;
        if (_dest_node) delete _dest_node;
        GpuFx::teardown();
    }
};

class FxConvFd1c1 : public FxConvFd {
   private:
    hipfftHandle _plan_r2c;
    hipfftHandle _plan_c2r;

    float* _ir_td = nullptr;
    hipfftComplex* _ir_fft = nullptr;
    float* _sig_td = nullptr;
    hipfftComplex* _sig_fft = nullptr;
    float* _wet_td = nullptr;
    float* _residual_td = nullptr;

    void allocateBuffers() {
        gpuErrChk(hipMalloc(&_ir_byte_buf, sizeof(char) * _ir_signal->getByteCount()));
        gpuErrChk(hipMalloc(&_ir_td, sizeof(float) * _fft_size));
        gpuErrChk(hipMalloc(&_ir_fft, sizeof(hipfftComplex) * _fft_size));
        gpuErrChk(hipMalloc(&_sig_td, sizeof(float) * _fft_size));
        gpuErrChk(hipMalloc(&_sig_fft, sizeof(hipfftComplex) * _fft_size));
        gpuErrChk(hipMalloc(&_wet_td, sizeof(float) * _fft_size));
        gpuErrChk(hipMalloc(&_residual_td, sizeof(float) * _fft_size));

        hipfftPlan1d(&_plan_r2c, _fft_size, HIPFFT_R2C, 1);
        hipfftPlan1d(&_plan_c2r, _fft_size, HIPFFT_C2R, 1);
    }

    void deallocateBuffers() {
        gpuErrChk(hipFree(_ir_byte_buf));
        gpuErrChk(hipFree(_ir_td));
        gpuErrChk(hipFree(_ir_fft));
        gpuErrChk(hipFree(_sig_td));
        gpuErrChk(hipFree(_sig_fft));
        gpuErrChk(hipFree(_wet_td));
        gpuErrChk(hipFree(_residual_td));

        hipfftDestroy(_plan_r2c);
        hipfftDestroy(_plan_c2r);
    }

    hipStream_t _process(hipStream_t stream, float* dst, const float* src, hipStreamCaptureStatus capture_status) override {
        // copy input as contiguous _n_proc_frames * float real signal into complex buffer to enable in-place fft
        IMemCpyNode::launchOrRecord1D(_sig_fft, src, sizeof(float), _n_proc_frames, hipMemcpyDeviceToDevice, stream, _src_node, capture_status);

        hipfftSetStream(_plan_r2c, stream);
        hipfftExecR2C(_plan_r2c, (hipfftReal*)_sig_fft, _sig_fft);

        // Convolution (Colplex Multiplication in Frequency Domain) (scaling is needed to retain unity gain in time domain after inverse fft)
        ccc_complexMultiplyAndScale<<<4, 768, 0, stream>>>(_sig_fft, _sig_fft, _ir_fft, _fft_size_non_redundant, 1.0f / _fft_size);

        // Inverse FFT
        hipfftSetStream(_plan_c2r, stream);
        // using inplace transform to avoid allocating additional buffers
        // output is _fft_size * float contiguous real signal, with the other half of the complex buffer being irrelevant
        hipfftExecC2R(_plan_c2r, _sig_fft, (hipfftReal*)_sig_fft);

        if (_force_wet_mix) {
            // if the fx should always produce a 100% wet output (e.g. amp cab ir), we can skip the mixing step and write the sum of the output and residual directly to the dst buffer
            IKernelNode::launchOrRecord(1, _n_proc_frames, 0, (void*)fff_pointwiseAdd, new void*[4]{&dst, &_sig_fft, &_residual_td, &_n_proc_frames}, stream, _dest_node, capture_status);
        } else {
            // combine convolution output with residual and write to dst buffer
            fff_pointwiseAdd<<<1, _n_proc_frames, 0, stream>>>(_wet_td, (float*)_sig_fft, _residual_td, _n_proc_frames);
            // mix dry and wet signal and write to dst buffer
            IKernelNode::launchOrRecord(1, _n_proc_frames, 0, (void*)fff_mix, new void*[5]{&dst, &src, &_wet_td, &_n_proc_frames, &_mix_ratio}, stream, _dest_node, capture_status);
        }
        return stream;
    }

   public:
    FxConvFd1c1(IPCMSignal* ir_signal, size_t max_ir_size, int ir_db_scale, bool force_wet_mix) : FxConvFd("FxConvFd1c1", ir_signal, max_ir_size, ir_db_scale, force_wet_mix) {
        if (_ir_signal->getChannelCount() > 1) {
            spdlog::warn("IR given to {} file is not mono. Only the first channel will be used.", _name);
        }
        _n_in_channels = 1;
        _n_out_channels = 1;
    }

    ~FxConvFd1c1() {}

    hipStream_t setup(hipStream_t stream, hipStreamCaptureStatus capture_status) override {
        GpuFx::setup(stream, capture_status);
        gpuErrChk(hipMemsetAsync(_residual_td, 0, sizeof(float) * _fft_size, stream));
        gpuErrChk(hipMemsetAsync(_ir_td, 0, sizeof(float) * _fft_size, stream));
        // gpuErrChk(hipMemsetAsync(_sig_td, 0, sizeof(float) * _fft_size, stream));
        gpuErrChk(hipMemsetAsync(_sig_fft, 0, sizeof(float2) * _fft_size, stream));

        // Memcopy2D is used since to copy only the first channel from multichannel IRs
        gpuErrChk(hipMemcpy2DAsync(_ir_byte_buf, _ir_signal->getByteDepth(), _ir_signal->getDataPtrConst(), _ir_signal->getByteDepth() * _ir_signal->getChannelCount(), _ir_signal->getByteDepth(), _ir_signal->getFrameCount(), hipMemcpyHostToDevice, stream));

        pcm_to_float_interleaved(_ir_td, _ir_byte_buf, getMinIROrFFTSize(), 1, _ir_signal->getBitDepthValue(), stream);
        hipfftSetStream(_plan_r2c, stream);
        hipfftExecR2C(_plan_r2c, (hipfftReal*)_ir_td, _ir_fft);

        // irs are usually normalized to max amplitude and therefore much too loud. We scale them down to avoid clipping
        f2_scale<<<_nBlocks, _nThreads, _nSharedMem, stream>>>(_ir_fft, _fft_size, getIRAttenuationFactor());
        return stream;
    }

    hipStream_t postProcess(hipStream_t stream, hipStreamCaptureStatus capture_status) override {
        // add convolution output to residual buffer and clear the unused residual tail
        // can't use inplace processing for rewinding, because when kernel is strided (thread count < fft size) or when using memcopy, the out-of-order execution of the threads could cause recursive reads of already processed data
        // which is why I'm missusing the _wet_td buffer as a temporary buffer for the sum of the output and residual
        // fff_pointwiseAdd<<<4, 768, 0, stream>>>(_wet_td, _residual_td + _n_proc_frames, _sig_td + _n_proc_frames, _fft_size - _n_proc_frames);
        fff_pointwiseAdd<<<4, 768, 0, stream>>>(_wet_td, _residual_td + _n_proc_frames, ((float*)_sig_fft) + _n_proc_frames, _fft_size - _n_proc_frames);
        gpuErrChk(hipMemcpyAsync(_residual_td, _wet_td, sizeof(float) * (_fft_size - _n_proc_frames), hipMemcpyDeviceToDevice, stream));
        gpuErrChk(hipMemsetAsync(_residual_td + _fft_size - _n_proc_frames, 0, sizeof(float) * _n_proc_frames, stream));

        // clear the part of _sig_fft that is not overwritten by the input
        // gpuErrChk(hipMemsetAsync(_sig_td + _n_proc_frames, 0, sizeof(float) * (_fft_size - _n_proc_samples), stream));
        gpuErrChk(hipMemsetAsync(_sig_fft, 0, sizeof(float2) * _fft_size, stream));

        return stream;
    }

    GpuFx* clone() override {
        return new FxConvFd1c1(_ir_signal->clone(), _fft_size, _ir_db_scale, _force_wet_mix);
    }
};

IGpuFx* IGpuFx::createConv1i1(IPCMSignal* ir_signal, size_t max_ir_size, int ir_db_scale, bool force_wet_mix) {
    return new FxConvFd1c1(ir_signal, max_ir_size, ir_db_scale, force_wet_mix);
}

class FxConvFd2c1 : public FxConvFd {
   private:
    hipfftHandle _plan_c2c;

    float* _ir_mono = nullptr;
    float2* _ir_stereo = nullptr;
    hipfftComplex* _ir_fft = nullptr;

    hipfftComplex* _sig_fft = nullptr;
    float2* _residual = nullptr;
    float2* _wet = nullptr;

    void allocateBuffers() {
        gpuErrChk(hipMalloc(&_ir_byte_buf, sizeof(char) * _ir_signal->getByteCount()));
        gpuErrChk(hipMalloc(&_ir_mono, sizeof(float) * getMinIROrFFTSize()));
        gpuErrChk(hipMalloc(&_ir_stereo, sizeof(float2) * _fft_size));
        gpuErrChk(hipMalloc(&_ir_fft, sizeof(hipfftComplex) * _fft_size));
        gpuErrChk(hipMalloc(&_sig_fft, sizeof(hipfftComplex) * _fft_size));
        gpuErrChk(hipMalloc(&_residual, sizeof(float2) * _fft_size));
        gpuErrChk(hipMalloc(&_wet, sizeof(float2) * _fft_size));

        hipfftPlan1d(&_plan_c2c, _fft_size, HIPFFT_C2C, 1);
    }

    void deallocateBuffers() {
        gpuErrChk(hipFree(_ir_byte_buf));
        gpuErrChk(hipFree(_ir_mono));
        gpuErrChk(hipFree(_ir_stereo));
        gpuErrChk(hipFree(_ir_fft));
        gpuErrChk(hipFree(_sig_fft));
        gpuErrChk(hipFree(_residual));
        gpuErrChk(hipFree(_wet));

        hipfftDestroy(_plan_c2c);
    }

    hipStream_t _process(hipStream_t stream, float* dst, const float* src, hipStreamCaptureStatus capture_status) {
        hipfftSetStream(_plan_c2c, stream);

        // pack stereo channels into real and img part of complex struct and perform fft both simultaneously
        // https://web.archive.org/web/20180312110051/http://www.engineeringproductivitytools.com/stuff/T0001/PT10.HTM
        // since the type hipfftComplex is a struct with two float members, we can simply copy our float2 buffer into the hipfftComplex buffer
        IMemCpyNode::launchOrRecord1D(_sig_fft, src, sizeof(float2), _n_proc_frames, hipMemcpyDeviceToDevice, stream, _src_node, capture_status);
        hipfftExecC2C(_plan_c2c, _sig_fft, _sig_fft, HIPFFT_FORWARD);

        // Convolution (Colplex Multiplication in Frequency Domain) (scaling is needed to retain unity gain in time domain after inverse fft)
        // According to the following link, when using the 2for1 method and convolving both signals with the same IR, a simple FLOAT multiplication of the respective real (ch1) and imag (ch2) parts is sufficient
        // https://web.archive.org/web/20180312110051/http://www.engineeringproductivitytools.com/stuff/T0001/PT10.HTM
        f2f2f2_multiplyAndScale<<<4, 768, 0, stream>>>(_sig_fft, _sig_fft, _ir_fft, _fft_size, 1.0f / _fft_size);

        // Inverse FFT
        hipfftExecC2C(_plan_c2c, _sig_fft, _sig_fft, HIPFFT_BACKWARD);

        if (_force_wet_mix) {
            // if the fx should always produce a 100% wet output (e.g. amp cab ir), we can skip the mixing step and write the sum of the output and residual directly to the dst buffer
            IKernelNode::launchOrRecord(1, _n_proc_frames, 0, (void*)f2f2f2_pointwiseAdd, new void*[4]{&dst, &_sig_fft, &_residual, &_n_proc_frames}, stream, _dest_node, capture_status);
        } else {
            // combine convolution output with residual and write to dst buffer
            f2f2f2_pointwiseAdd<<<1, _n_proc_frames, 0, stream>>>(_wet, _sig_fft, _residual, _n_proc_frames);
            // mix dry and wet signal and write to dst buffer
            IKernelNode::launchOrRecord(1, _n_proc_frames, 0, (void*)f2f2f2_mix, new void*[5]{&dst, &src, &_wet, &_n_proc_frames, &_mix_ratio}, stream, _dest_node, capture_status);
        }

        return stream;
    }

   public:
    FxConvFd2c1(IPCMSignal* ir_signal, size_t max_ir_size, int ir_db_scale, bool force_wet_mix) : FxConvFd("FxConvFd2c1", ir_signal, max_ir_size, ir_db_scale, force_wet_mix) {
        if (_ir_signal->getChannelCount() > 1) {
            spdlog::warn("IR file given to {} is not mono. Only the first channel will be used.", _name);
        }
        _n_in_channels = 2;
        _n_out_channels = 2;
    }

    ~FxConvFd2c1() {
    }

    hipStream_t setup(hipStream_t stream, hipStreamCaptureStatus capture_status) override {
        GpuFx::setup(stream, capture_status);
        gpuErrChk(hipMemsetAsync(_residual, 0, sizeof(float2) * _fft_size, stream));
        gpuErrChk(hipMemsetAsync(_ir_stereo, 0, sizeof(float2) * _fft_size, stream));
        gpuErrChk(hipMemsetAsync(_sig_fft, 0, sizeof(hipfftComplex) * _fft_size, stream));

        // copy ir to device and convert to float. Memcopy2D is used since to copy only the first channel from multichannel IRs
        gpuErrChk(hipMemcpy2DAsync(_ir_byte_buf, _ir_signal->getByteDepth(), _ir_signal->getDataPtrConst(), _ir_signal->getByteDepth() * _ir_signal->getChannelCount(), _ir_signal->getByteDepth(), _ir_signal->getFrameCount(), hipMemcpyHostToDevice, stream));
        // here we copy the mono IR to stereo and then cast it to complex buffer to perform an FFT on both channels simultaneously
        // while this seems to unnecessarily redundant, this theqnique allows us to multiply the IR FFT with a stereo signal FFT that is optained the same way
        // as a result, each processing pass only requires one forware and one inverse FFT instead of two for each direction
        // https://web.archive.org/web/20180312110051/http://www.engineeringproductivitytools.com/stuff/T0001/PT10.HTM
        pcm_to_float_interleaved(_ir_mono, _ir_byte_buf, getMinIROrFFTSize(), 1, _ir_signal->getBitDepthValue(), stream);

        // TODO: produces invalid argument error for the second fx instance in a signal graph. replace 2D copies with multi memcpy once fixed
        // IMemCpyNode::launchOrRecordMulti(MultiMemcpyType::Segmented2Interleaved, _ir_stereo, &_ir_mono, sizeof(float), getMinIROrFFTSize(), 2, {0,0}, hipMemcpyDeviceToDevice, stream, nullptr, capture_status);
        gpuErrChk(hipMemcpy2DAsync(_ir_stereo, sizeof(float2), _ir_mono, sizeof(float), sizeof(float), getMinIROrFFTSize(), hipMemcpyDeviceToDevice, stream));
        gpuErrChk(hipMemcpy2DAsync(((float*)_ir_stereo) + 1, sizeof(float2), _ir_mono, sizeof(float), sizeof(float), getMinIROrFFTSize(), hipMemcpyDeviceToDevice, stream));

        hipfftSetStream(_plan_c2c, stream);
        hipfftExecC2C(_plan_c2c, (hipfftComplex*)_ir_stereo, _ir_fft, HIPFFT_FORWARD);

        // irs are usually normalized to max amplitude and therefore much too loud. We scale them down to avoid clipping
        f2_scale<<<4, 768, 0, stream>>>(_ir_fft, _fft_size, getIRAttenuationFactor());
        return stream;
    }

    hipStream_t postProcess(hipStream_t stream, hipStreamCaptureStatus capture_status) override {
        // add convolution output to residual buffer and clear the unused residual tail
        // can't use inplace processing for rewinding, because when kernel is strided (thread count < fft size) or when using memcopy, the out-of-order execution of the threads could cause recursive reads of already processed data
        // which is why I'm missusing the _wet buffer as a temporary buffer for the sum of the output and residual
        f2f2f2_pointwiseAdd<<<4, 768, 0, stream>>>(_wet, _residual + _n_proc_frames, _sig_fft + _n_proc_frames, _fft_size - _n_proc_frames);
        gpuErrChk(hipMemcpyAsync(_residual, _wet, sizeof(float2) * (_fft_size - _n_proc_frames), hipMemcpyDeviceToDevice, stream));
        gpuErrChk(hipMemsetAsync(_residual + _fft_size - _n_proc_frames, 0, sizeof(float2) * _n_proc_frames, stream));

        // clear the part of _sig_fft that is not overwritten by the input
        gpuErrChk(hipMemsetAsync(_sig_fft + _n_proc_frames, 0, sizeof(float2) * (_fft_size - _n_proc_frames), stream));
        return stream;
    }

    GpuFx* clone() override {
        return new FxConvFd2c1(_ir_signal->clone(), _fft_size, _ir_db_scale, _force_wet_mix);
    }
};
IGpuFx* IGpuFx::createConv2i1(IPCMSignal* ir_signal, size_t max_ir_size, int ir_db_scale, bool force_wet_mix) {
    return new FxConvFd2c1(ir_signal, max_ir_size, ir_db_scale, force_wet_mix);
}

class FxConvFd2c2 : public FxConvFd {
   private:
    hipfftHandle _plan_c2c;
    hipfftHandle _plan_c2r;

    float2* _residual;
    float2* _rirBuf;
    float2* _output;
    float2* _wet;
    float* _left;
    float* _right;
    hipfftComplex* _rir_fft_packed;
    hipfftComplex* _sig_fft_packed;
    hipStream_t _stream_right;
    struct
    {
        hipfftComplex *left, *right;
    } _sigFFT, _rirFFT;

    float2** float2_buffers[9] = {
        &_rir_fft_packed,
        &_rirFFT.left,
        &_rirFFT.right,
        &_sig_fft_packed,
        &_sigFFT.left,
        &_sigFFT.right,
        &_output,
        &_wet,
        &_residual,
    };

    void allocateBuffers() {
        for (size_t i = 0; i < sizeof(float2_buffers) / sizeof(*float2_buffers); i++) {
            gpuErrChk(hipMalloc(float2_buffers[i], _fft_size * sizeof(*float2_buffers)));
        }
        gpuErrChk(hipMalloc(&_ir_byte_buf, sizeof(char) * _ir_signal->getByteCount()));
        hipfftPlan1d(&_plan_c2c, _fft_size, HIPFFT_C2C, 1);

        hipfftPlan1d(&_plan_c2r, _fft_size, HIPFFT_C2R, 1);
        gpuErrChk(hipStreamCreate(&_stream_right));
    }

    void deallocateBuffers() {
        for (size_t i = 0; i < sizeof(float2_buffers) / sizeof(*float2_buffers); i++) {
            gpuErrChk(hipFree(*float2_buffers[i]));
        }
        gpuErrChk(hipFree(_ir_byte_buf));
        hipfftDestroy(_plan_c2c);
        hipfftDestroy(_plan_c2r);
        gpuErrChk(hipStreamDestroy(_stream_right));
    }

    hipStream_t _process(hipStream_t stream, float* dst, const float* src, hipStreamCaptureStatus capture_status) override {
        hipfftSetStream(_plan_c2c, stream);
        hipfftSetStream(_plan_c2r, stream);

        // pack stereo channels into real and img part of complex struct and perform fft both simultaneously
        // https://web.archive.org/web/20180312110051/http://www.engineeringproductivitytools.com/stuff/T0001/PT10.HTM
        // since the type hipfftComplex is a struct with two float members, we can simply copy our float2 buffer into the hipfftComplex buffer
        IMemCpyNode::launchOrRecord1D(_sig_fft_packed, src, sizeof(float2), _n_proc_frames, hipMemcpyDeviceToDevice, stream, _src_node, capture_status);
        hipfftExecC2C(_plan_c2c, _sig_fft_packed, _sig_fft_packed, HIPFFT_FORWARD);
        ccc_unpackCto2C<<<4, 768, 0, stream>>>(_sigFFT.left, _sigFFT.right, _sig_fft_packed, _fft_size);

        // Convolution (Colplex Multiplication in Frequency Domain) (scaling is needed to retain unity gain in time domain after inverse fft)
        ccc_complexMultiplyAndScale<<<4, 768, 0, stream>>>(_sigFFT.left, _sigFFT.left, _rirFFT.left, _fft_size_non_redundant, 1.0f / _fft_size);
        ccc_complexMultiplyAndScale<<<4, 768, 0, stream>>>(_sigFFT.right, _sigFFT.right, _rirFFT.right, _fft_size_non_redundant, 1.0f / _fft_size);

        // Inverse FFT
        // using inplace transform to avoid allocating additional buffers
        // output is _fft_size * float contiguous real signal, with the other half of the complex buffer being irrelevant
        hipfftExecC2R(_plan_c2r, _sigFFT.left, (hipfftReal*)_sigFFT.left);
        hipfftExecC2R(_plan_c2r, _sigFFT.right, (hipfftReal*)_sigFFT.right);

        if (_force_wet_mix) {
            // if the fx should always produce a 100% wet output (e.g. amp cab ir), we can skip the mixing step and write the sum of the output and residual directly to the dst buffer
            IKernelNode::launchOrRecord(1, _n_proc_frames, 0, (void*)f2fff2_pointwiseAdd, new void*[5]{&dst, &_sigFFT.left, &_sigFFT.right, &_residual, &_n_proc_frames}, stream, _dest_node, capture_status);
        } else {
            // combine convolution output with residual and write to dst buffer
            f2fff2_pointwiseAdd<<<1, _n_proc_frames, 0, stream>>>(_wet, (float*)_sigFFT.left, (float*)_sigFFT.right, _residual, _n_proc_frames);
            // mix dry and wet signal and write to dst buffer
            IKernelNode::launchOrRecord(1, _n_proc_frames, 0, (void*)f2f2f2_mix, new void*[5]{&dst, &src, &_wet, &_n_proc_frames, &_mix_ratio}, stream, _dest_node, capture_status);
        }

        return stream;
    }

   public:
    FxConvFd2c2(IPCMSignal* ir_signal, size_t max_ir_size, int ir_db_scale, bool force_wet_mix) : FxConvFd("FxConvFd2c2", ir_signal, max_ir_size, ir_db_scale, force_wet_mix) {
        if (_ir_signal->getChannelCount() < 2) {
            throw std::runtime_error("IR file given to {} has less than two channels. Stereo IRs are required for this fx.");
        } else if (_ir_signal->getChannelCount() > 2) {
            spdlog::warn("IR file given to {} is not stereo. Only the first two channels will be used.", _name);
        }
        _n_in_channels = 2;
        _n_out_channels = 2;
    }

    ~FxConvFd2c2() {}

    hipStream_t setup(hipStream_t stream, hipStreamCaptureStatus capture_status) override {
        GpuFx::setup(stream, capture_status);
        gpuErrChk(hipMemsetAsync(_ir_byte_buf, 0, sizeof(char) * _ir_signal->getByteCount(), stream));
        gpuErrChk(hipMemsetAsync(_residual, 0, sizeof(float2) * _fft_size, stream));
        gpuErrChk(hipMemsetAsync(_rir_fft_packed, 0, sizeof(hipfftComplex) * _fft_size, stream));
        gpuErrChk(hipMemsetAsync(_sig_fft_packed, 0, sizeof(hipfftComplex) * _fft_size, stream));

        // TODO: Remove and fix fft buffer sizes
        gpuErrChk(hipMemsetAsync(_rirFFT.left, 0, sizeof(hipfftComplex) * _fft_size, stream));
        gpuErrChk(hipMemsetAsync(_rirFFT.right, 0, sizeof(hipfftComplex) * _fft_size, stream));

        gpuErrChk(hipMemcpyAsync(_ir_byte_buf, _ir_signal->getDataPtrConst(), sizeof(char) * _ir_signal->getByteCount(), hipMemcpyHostToDevice, stream));
        // pack stereo channels into real and img part of complex struct and perform fft both simultaneously
        // https://web.archive.org/web/20180312110051/http://www.engineeringproductivitytools.com/stuff/T0001/PT10.HTM
        pcm_to_float2((float2*)_rir_fft_packed, _ir_byte_buf, getMinIROrFFTSize(), _ir_signal->getChannelCount(), _ir_signal->getBitDepthValue(), stream);
        hipfftSetStream(_plan_c2c, stream);
        hipfftExecC2C(_plan_c2c, _rir_fft_packed, _rir_fft_packed, HIPFFT_FORWARD);
        ccc_unpackCto2C<<<4, 768, 0, stream>>>(_rirFFT.left, _rirFFT.right, _rir_fft_packed, _fft_size);

        // irs are usually normalized to max amplitude and therefore much too loud. We scale them down to avoid clipping
        f2_scale<<<4, 768, 0, stream>>>(_rirFFT.left, _fft_size, getIRAttenuationFactor());
        f2_scale<<<4, 768, 0, stream>>>(_rirFFT.right, _fft_size, getIRAttenuationFactor());
        gpuErrChk(hipMemsetAsync(_residual, 0, sizeof(float2) * _fft_size, stream));
        return stream;
    }

    hipStream_t postProcess(hipStream_t stream, hipStreamCaptureStatus capture_status) override {
        // add convolution output to residual buffer and clear the unused residual tail
        // can't use inplace processing for rewinding, because when kernel is strided (thread count < fft size) or when using memcopy, the out-of-order execution of the threads could cause recursive reads of already processed data
        // which is why I'm missusing the _wet buffer as a temporary buffer for the sum of the output and residual
        f2fff2_pointwiseAdd<<<4, 768, 0, stream>>>(_wet, ((float*)_sigFFT.left) + _n_proc_frames, ((float*)_sigFFT.right) + _n_proc_frames, _residual + _n_proc_frames, _fft_size - _n_proc_frames);
        gpuErrChk(hipMemcpyAsync(_residual, _wet, sizeof(float2) * _fft_size - _n_proc_frames, hipMemcpyDeviceToDevice, stream));
        gpuErrChk(hipMemsetAsync(_residual + _fft_size - _n_proc_frames, 0, sizeof(float2) * _n_proc_frames, stream));

        // clear the part of _sig_fft_packed that is not overwritten by the input
        gpuErrChk(hipMemsetAsync(_sig_fft_packed + _n_proc_frames, _nSharedMem, sizeof(hipfftComplex) * (_fft_size - _n_proc_frames), stream));
        return stream;
    }

    GpuFx* clone() override {
        return new FxConvFd2c2(_ir_signal->clone(), _fft_size, _ir_db_scale, _force_wet_mix);
    }
};

IGpuFx* IGpuFx::createConv2i2(IPCMSignal* ir_signal, size_t max_ir_size, int ir_db_scale, bool force_wet_mix) {
    return new FxConvFd2c2(ir_signal, max_ir_size, ir_db_scale, force_wet_mix);
}