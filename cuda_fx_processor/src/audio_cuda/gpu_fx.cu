#include <cuda_ext.cuh>

#include "gpu_fx.cuh"
#include "spdlog/spdlog.h"

class GpuFx : public IGpuFx {
   private:
    hipGraph_t _setup_graph = nullptr;
    hipGraph_t _process_graph = nullptr;
    hipGraph_t _post_process_graph = nullptr;
    bool _destroy_setup_graph = false;
    bool _destroy_process_graph = false;
    bool _destroy_post_process_graph = false;
    bool _has_post_processing;

    hipStream_t getRecordingStream() {
        hipStream_t stream;
        gpuErrChk(hipStreamCreate(&stream));
        gpuErrChk(hipStreamBeginCapture(stream, hipStreamCaptureMode::hipStreamCaptureModeGlobal));
        return stream;
    }
    hipGraph_t recordStreamToGraph(hipStream_t stream, hipGraph_t* graph) {
        gpuErrChk(hipGraphCreate(graph, 0));
        gpuErrChk(hipStreamEndCapture(stream, graph));
        gpuErrChk(hipStreamDestroy(stream));
        return *graph;
    }
    hipGraph_t createGraphWithEmpyNode(hipGraph_t* graph) {
        gpuErrChk(hipGraphCreate(graph, 0));
        hipGraphNode_t emptyNode;
        gpuErrChk(hipGraphAddEmptyNode(&emptyNode, *graph, nullptr, 0));
        return *graph;
    }

   protected:
    std::string _name;
    size_t _n_in_channels;
    size_t _n_out_channels;
    size_t _n_proc_channels;
    size_t _n_proc_frames;
    size_t _n_proc_samples;
    size_t _mix_ratio_param_index;
    float _mix_ratio;
    bool _has_soft_param_update = false;

    BufferRackSpecs _input_specs;
    BufferRackSpecs _output_specs;

    IKernelNode* _mix_node=nullptr;

    virtual void allocateBuffers() = 0;
    virtual void deallocateBuffers() = 0;
    virtual void setMixRatio(float mix_ratio, int kernel_param_index = -1) {
        if (mix_ratio < 0.0f || mix_ratio > 1.0f) {
            throw std::runtime_error("Mix ratio must be between 0.0 and 1.0");
        } else if (mix_ratio == _mix_ratio) {
            return;
        }
        _mix_ratio = mix_ratio;
        _mix_node->updateKernelParamAt(kernel_param_index >= 0 ? kernel_param_index : _mix_ratio_param_index, &_mix_ratio);
        _has_soft_param_update = true;
    }

   public:
    GpuFx(std::string name, bool has_post_processing = true, float mix_ratio = 1.0f) : _name(name), _has_post_processing(has_post_processing), _mix_ratio(mix_ratio), _mix_ratio_param_index(4) {}
    virtual ~GpuFx() {
        delete _mix_node;
    };

    std::string getName() {
        return _name;
    }
    size_t getInChannelCount() { return _n_in_channels; }
    size_t getOutChannelCount() { return _n_out_channels; }
    size_t getProcSampleCount() { return _n_proc_samples; }
    size_t getOutSampleCount() { return _n_proc_frames * _n_out_channels; }

    BufferRackSpecs getInputSpecs() { return _input_specs; }
    BufferRackSpecs getOutputSpecs() { return _output_specs; }

    virtual void configure(size_t n_proc_frames, size_t n_in_channels = 0, size_t n_out_channels = 0) {
        if (n_out_channels == 0 || n_in_channels == 0) {
            throw std::runtime_error("n_in_channels " + std::to_string(n_in_channels) + " and n_out_channels " + std::to_string(n_out_channels) + " must be greater than 0");
        }
        _n_proc_frames = n_proc_frames;
        _n_in_channels = n_in_channels;
        _n_out_channels = n_out_channels;
        _n_proc_channels = n_in_channels;
        _n_proc_samples = _n_proc_frames * _n_proc_channels;
        _input_specs = BufferRackSpecs(BufferSpecs(MemoryContext::DEVICE, _n_proc_frames, _n_in_channels, ChannelOrder::INTERLEAVED));
        _output_specs = BufferRackSpecs(BufferSpecs(MemoryContext::DEVICE, _n_proc_frames, _n_out_channels, ChannelOrder::INTERLEAVED));
    }
    virtual hipGraph_t recordSetupGraph() {
        allocateBuffers();
        _destroy_setup_graph = true;
        return recordStreamToGraph(setup(getRecordingStream(), hipStreamCaptureStatus::hipStreamCaptureStatusActive), &_setup_graph);
    }

    virtual hipGraph_t recordProcessGraph(const BufferRack* dest, const BufferRack* src) {
        _destroy_process_graph = true;
        return recordStreamToGraph(process(getRecordingStream(), dest, src, hipStreamCaptureStatus::hipStreamCaptureStatusActive), &_process_graph);
    }

    virtual hipGraph_t recordPostProcessGraph() {
        _destroy_post_process_graph = true;
        if (_has_post_processing) {
            return recordStreamToGraph(postProcess(getRecordingStream(), hipStreamCaptureStatus::hipStreamCaptureStatusActive), &_post_process_graph);
        } else {
            return createGraphWithEmpyNode(&_post_process_graph);
        }
    }

    virtual hipStream_t setup(hipStream_t stream, hipStreamCaptureStatus capture_status = hipStreamCaptureStatus::hipStreamCaptureStatusNone) {
        if (capture_status == hipStreamCaptureStatus::hipStreamCaptureStatusNone) {
            allocateBuffers();
        }
        return stream;
    }
    virtual hipStream_t process(hipStream_t stream, const BufferRack* dest, const BufferRack* src, hipStreamCaptureStatus capture_status = hipStreamCaptureStatus::hipStreamCaptureStatusNone) = 0;
    virtual hipStream_t postProcess(hipStream_t stream, hipStreamCaptureStatus capture_status = hipStreamCaptureStatus::hipStreamCaptureStatusNone) { return stream; }
    virtual void setSoftParams(float mix_ratio) {
        setMixRatio(mix_ratio);
    }
    virtual void updateSoftParams(hipGraphExec_t proc_graph_node, hipGraphNode_t child_graph_node) {
        if (_has_soft_param_update) {
            gpuErrChk(hipGraphExecChildGraphNodeSetParams(proc_graph_node, child_graph_node, _process_graph));
            _has_soft_param_update = false;
        }
    };
    virtual void updateBufferPtrs(hipGraphExec_t procGraphExec, const BufferRack* dst, const BufferRack* src) { throw std::runtime_error("Not implemented"); };
    virtual void teardown() {
        deallocateBuffers();
        if (_destroy_setup_graph) gpuErrChk(hipGraphDestroy(_setup_graph));
        if (_destroy_process_graph) gpuErrChk(hipGraphDestroy(_process_graph));
        if (_destroy_post_process_graph) gpuErrChk(hipGraphDestroy(_post_process_graph));
    }
    virtual GpuFx* clone() {
        throw std::runtime_error("Not implemented");
    }
};