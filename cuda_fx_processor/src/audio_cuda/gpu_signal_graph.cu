#include "hip/hip_runtime.h"

#include <errno.h>
#include <sys/resource.h>
#include <unistd.h>

#include <atomic>
#include <cuda_ext.cuh>
#include <future>
#include <log.hpp>
#include <stdexcept>
#include <thread>

#include "block_buffer.cuh"
#include "gpu.cuh"
#include "gpu_signal_graph.cuh"
#include "spdlog/spdlog.h"

class GpuSignalVertex : public IGpuSignalVertex {
   protected:
    bool _owns_dest_buffer;
    BufferRack _src_ptr;
    BufferRack _dest_ptr;

    std::vector<IGpuSignalVertex*> _parents;
    std::vector<IGpuSignalVertex*> _children;

    void addToGraph(hipGraph_t setup_graph, hipGraph_t process_graph) {
        std::vector<hipGraphNode_t> dependencies;
        for (IGpuSignalVertex* parent : _parents) {
            dependencies.push_back(parent->getProcessNode());
        }
        addToGraph(setup_graph, process_graph, dependencies.data(), dependencies.size());
    }

    virtual void addToGraph(hipGraph_t setup_graph, hipGraph_t process_graph, hipGraphNode_t* dependencies, size_t n_dependencies) = 0;

   public:
    GpuSignalVertex(IGpuSignalVertex* parent, std::vector<Buffer*> src_ptr = {}, std::vector<Buffer*> dst_ptr = {}) : _parents(parent ? std::vector<IGpuSignalVertex*>{parent} : std::vector<IGpuSignalVertex*>{}), _src_ptr(src_ptr), _dest_ptr(dst_ptr) {
    }
    GpuSignalVertex(std::vector<IGpuSignalVertex*> parents, std::vector<Buffer*> src_ptr = {}, std::vector<Buffer*> dst_ptr = {}) : _parents(parents), _src_ptr(src_ptr), _dest_ptr(dst_ptr) {
    }

    ~GpuSignalVertex() {
    }

    virtual const BufferRackSpecs& getInputSpecs() { return _src_ptr.getSpecs(); }
    virtual const BufferRackSpecs& getOutputSpecs() { return _dest_ptr.getSpecs(); }
    virtual void setSrcPtr(std::vector<Buffer*> src_ptr, hipGraphExec_t graph_exec) = 0;
    virtual void setDestPtr(std::vector<Buffer*> dest_ptr, hipGraphExec_t graph_exec) = 0;
    std::vector<IGpuSignalVertex*>& getParents() override { return _parents; }
    std::vector<IGpuSignalVertex*>& getChildren() override { return _children; }
    virtual void setup(hipGraph_t setup_graph, hipGraph_t process_graph, size_t n_proc_frames, std::vector<Buffer*>& buffer_ptrs) = 0;
};

class GpuSignalCopyVertex : public GpuSignalVertex {
   private:
    bool _owns_dest_buffer;
    size_t _n_channels;
    IMemCpyNode* _node;

    void addToGraph(hipGraph_t setup_graph, hipGraph_t process_graph) {
        std::vector<hipGraphNode_t> dependencies;
        for (IGpuSignalVertex* parent : _parents) {
            dependencies.push_back(parent->getProcessNode());
        }
        addToGraph(setup_graph, process_graph, dependencies.data(), dependencies.size());
    }

    void addToGraph(hipGraph_t setup_graph, hipGraph_t process_graph, hipGraphNode_t* dependencies, size_t n_dependencies) override {
        throw std::runtime_error("Not implemented");
    }

   public:
    GpuSignalCopyVertex(IMemCpyNode* node, std::vector<Buffer*> src_ptr, std::vector<Buffer*> dst_ptr, size_t n_channels, std::vector<IGpuSignalVertex*> parents = {}, std::vector<IGpuSignalVertex*> children = {})
        : GpuSignalVertex(parents, src_ptr, dst_ptr), _node(node), _n_channels(n_channels) {
    }
    ~GpuSignalCopyVertex() {
    }

    const std::vector<Buffer*>& getSrcPtr() override {
        return _src_ptr.getBuffers();
    }
    const std::vector<Buffer*>& getDestPtr() override {
        return _dest_ptr.getBuffers();
    }
    void setSrcPtr(const std::vector<Buffer*> src_ptr, hipGraphExec_t graph_exec) override {
        _src_ptr.set(src_ptr);
        _node->updateSrcPtr(_src_ptr.getDataMod(), graph_exec);
    }
    void setDestPtr(std::vector<Buffer*> dest_ptr, hipGraphExec_t graph_exec) override {
        _dest_ptr.set(dest_ptr);
        _node->updateDstPtr(_dest_ptr.getDataMod(), graph_exec);
    }
    hipGraphNode_t getProcessNode() override { return _node->getNode(); }
    hipGraphNode_t* getProcessNodePtr() override { return _node->getNodePtr(); }
    size_t getIncomingChannelCount() override { return _n_channels; }
    size_t getOutgoingChannelCount() override { return _n_channels; }

    void setup(hipGraph_t setup_graph, hipGraph_t process_graph, size_t n_proc_frames, std::vector<Buffer*>& buffer_ptrs) override {
        throw std::runtime_error("Not implemented");
    }
};

class GpuSignalFxVertex : public GpuSignalVertex {
   private:
    IGpuFx* _fx;
    hipGraphNode_t _setup_node;
    hipGraphNode_t _process_node;
    hipGraphNode_t _post_process_node;

    void addToGraph(hipGraph_t setup_graph, hipGraph_t process_graph, hipGraphNode_t* dependencies, size_t n_dependencies) override {
        hipGraphAddChildGraphNode(&_setup_node, setup_graph, nullptr, 0, _fx->recordSetupGraph());
        hipGraphAddChildGraphNode(&_process_node, process_graph, dependencies, n_dependencies, _fx->recordProcessGraph(&_dest_ptr, &_src_ptr));
        hipGraphAddChildGraphNode(&_post_process_node, process_graph, &_process_node, 1, _fx->recordPostProcessGraph());
    }

   public:
    GpuSignalFxVertex(IGpuFx* fx, IGpuSignalVertex* parent) : GpuSignalVertex(parent), _fx(fx) {
    }
    GpuSignalFxVertex(IGpuFx* fx, std::vector<IGpuSignalVertex*> parents) : GpuSignalVertex(parents), _fx(fx) {
    }

    ~GpuSignalFxVertex() {
    }

    IGpuFx* getFx() { return _fx; }
    const std::vector<Buffer*>& getSrcPtr() override { return _src_ptr.getBuffers(); }
    const std::vector<Buffer*>& getDestPtr() override { return _dest_ptr.getBuffers(); }
    void setSrcPtr(const std::vector<Buffer*> src_ptr, hipGraphExec_t graph_exec) override {
        _src_ptr.set(src_ptr);
        _fx->updateBufferPtrs(nullptr, &_dest_ptr, &_src_ptr);
    }
    void setDestPtr(std::vector<Buffer*> dest_ptr, hipGraphExec_t graph_exec) override {
        _dest_ptr.set(dest_ptr);
        _fx->updateBufferPtrs(nullptr, &_dest_ptr, &_src_ptr);
    }
    hipGraphNode_t getProcessNode() override { return _process_node; }
    hipGraphNode_t* getProcessNodePtr() override { return &_process_node; }

    size_t getIncomingChannelCount() override {
        return std::accumulate(_parents.begin(), _parents.end(), 0, [](size_t a, IGpuSignalVertex* b) { return a + b->getOutgoingChannelCount(); });
    }

    size_t getOutgoingChannelCount() override {
        return _fx->getOutChannelCount();
    }

    void setup(hipGraph_t setup_graph, hipGraph_t process_graph, size_t n_proc_frames, std::vector<Buffer*>& buffer_ptrs) {
        _fx->configure(n_proc_frames, getIncomingChannelCount());

        bool has_multiple_parents = _parents.size() > 1;
        bool has_parent_with_multiple_children = std::any_of(_parents.begin(), _parents.end(), [](IGpuSignalVertex* parent) { return parent->getChildren().size() > 1; });
        bool output_channel_counts_mismatch = _parents.front()->getOutgoingChannelCount() != getOutgoingChannelCount();

        if (has_multiple_parents) {
            BufferRackSpecs specs;
            std::vector<Buffer*> buffers;
            for (size_t i = 0; i < _parents.size(); i++) {
                buffers.insert(buffers.end(), _parents[i]->getDestPtr().begin(), _parents[i]->getDestPtr().end());
            }
            _src_ptr.set(buffers);
        } else {
            _src_ptr = _parents.front()->getDestPtr();
        }

        if (_children.empty() || has_multiple_parents || has_parent_with_multiple_children || output_channel_counts_mismatch) {
            // can not use in place processing -> allocate new destination buffer
            _dest_ptr.set(_fx->getOutputSpecs());
            buffer_ptrs.insert(buffer_ptrs.end(), _dest_ptr.getBuffers().begin(), _dest_ptr.getBuffers().end());
        } else {
            // use in place processing
            _dest_ptr.set(_src_ptr.getBuffers());
        }

        GpuSignalVertex::addToGraph(setup_graph, process_graph);
    }
};

class GpuSignalGraph : public IGpuSignalGraph {
   private:
    std::vector<GpuSignalVertex*> _roots;
    std::vector<GpuSignalVertex*> _vertices;
    std::vector<GpuSignalVertex*> _leaves;
    std::vector<Buffer*> _buffer_ptrs;
    std::vector<GpuSignalVertex*> _input_vertices;
    std::vector<GpuSignalVertex*> _output_vertices;

    size_t _n_proc_frames;
    size_t _n_in_channels;
    size_t _n_out_channels;

    hipGraph_t _setup_graph;
    hipGraph_t _process_graph;
    hipGraphExec_t _setup_graph_exec;
    hipGraphExec_t _process_graph_exec;
    hipStream_t _stream;

    void updateLeaves() {
        _leaves.clear();
        for (GpuSignalVertex* vertices : _vertices) {
            if (vertices->getChildren().empty()) {
                _leaves.push_back(vertices);
            }
        }
    }

   public:
    GpuSignalGraph() {
    }

    ~GpuSignalGraph() {
        for (GpuSignalVertex* vertex : _vertices) {
            delete vertex;
        }
        for (GpuSignalVertex* vertex : _input_vertices) {
            delete vertex;
        }
        for (GpuSignalVertex* vertex : _output_vertices) {
            delete vertex;
        }
        gpuErrChk(hipGraphDestroy(_process_graph));
        gpuErrChk(hipGraphExecDestroy(_process_graph_exec));
    }

    size_t getInputChannelCount() {
        return _n_in_channels;
    }

    size_t getOutputChannelCount() {
        return _n_out_channels;
    }

    void setup(size_t n_proc_frames, size_t n_in_channels, size_t n_out_channels) override {
        _n_proc_frames = n_proc_frames;
        _n_in_channels = n_in_channels;
        gpuErrChk(hipGraphCreate(&_setup_graph, 0));
        gpuErrChk(hipGraphCreate(&_process_graph, 0));

        std::vector<GpuSignalVertex*> queue;
        std::vector<GpuSignalVertex*> orphans;
        std::vector<IGpuSignalVertex*> orphans_i;
        std::copy_if(_vertices.begin(), _vertices.end(), std::back_inserter(orphans), [](GpuSignalVertex* vertex) { return vertex->getParents().empty(); });
        std::transform(orphans.begin(), orphans.end(), std::back_inserter(orphans_i), [](GpuSignalVertex* vertex) { return static_cast<GpuSignalFxVertex*>(vertex); });
        std::copy(_vertices.begin(), _vertices.end(), std::back_inserter(queue));

        for (size_t i = 0; i < _n_in_channels; i++) {
            BufferRack src_ptr(BufferSpecs(MemoryContext::HOST, n_proc_frames));
            BufferRack dest_ptr(BufferSpecs(MemoryContext::DEVICE, n_proc_frames));
            auto input_vertex = new GpuSignalCopyVertex(
                IMemCpyNode::create1D(dest_ptr.getDataMod(), src_ptr.getDataMod(), sizeof(float), _n_proc_frames, hipMemcpyHostToDevice, _process_graph),
                src_ptr.getBuffers(), dest_ptr.getBuffers(), 1, {}, orphans_i);
            src_ptr.deallocateBuffers();
            for (Buffer* buffer : dest_ptr.getBuffers()) {
                _buffer_ptrs.push_back(buffer);
            }
            _input_vertices.push_back(input_vertex);
            _roots.push_back(input_vertex);
        }

        for (GpuSignalVertex* orphan : orphans) {
            orphan->getParents().insert(orphan->getParents().end(), _input_vertices.begin(), _input_vertices.end());
        }

        std::vector<GpuSignalVertex*> visited = std::vector<GpuSignalVertex*>{_roots};
        size_t queue_index = 0;
        while (!queue.empty()) {
            auto vertex = queue.at(queue_index);
            if (std::all_of(vertex->getParents().begin(), vertex->getParents().end(), [&visited](IGpuSignalVertex* parent) { return std::find(visited.begin(), visited.end(), parent) != visited.end(); })) {
                vertex->setup(_setup_graph, _process_graph, _n_proc_frames, _buffer_ptrs);
                queue.erase(queue.begin() + queue_index);
                visited.push_back(vertex);
                queue_index = 0;
            } else {
                queue_index++;
            }
        }

        if (_leaves.empty()) {
            std::copy(_input_vertices.begin(), _input_vertices.end(), std::back_inserter(_leaves));
        }
        size_t n_leave_outputs = std::accumulate(_leaves.begin(), _leaves.end(), 0, [](size_t sum, GpuSignalVertex* v) { return sum + v->getOutgoingChannelCount(); });
        _n_out_channels = n_out_channels == 0 ? n_leave_outputs : n_out_channels;

        if (n_leave_outputs != _n_out_channels) {
            throw std::runtime_error("Leave nodes produce " + std::to_string(n_leave_outputs) + " output channels, but " + std::to_string(_n_out_channels) + " are expected");
        } else {
            size_t i = 0;
            for (GpuSignalVertex* leave : _leaves) {
                for (size_t c = 0; c < leave->getOutgoingChannelCount(); c++) {
                    BufferRack dest_ptr(BufferSpecs(MemoryContext::HOST, n_proc_frames));
                    _output_vertices.push_back(
                        new GpuSignalCopyVertex(
                            IMemCpyNode::create1D(dest_ptr.getDataMod(), leave->getDestPtr()[c]->getDataMod(), sizeof(float), _n_proc_frames, hipMemcpyDeviceToHost, _process_graph, leave->getProcessNodePtr(), 1),
                            {leave->getDestPtr()[c]}, dest_ptr.getBuffers(), 1, {leave}, {}));
                    dest_ptr.deallocateBuffers();
                    i++;
                }
            }
        }

        gpuErrChk(hipStreamCreate(&_stream));
        gpuErrChk(hipGraphInstantiate(&_setup_graph_exec, _setup_graph, NULL, NULL, 0));
        gpuErrChk(hipGraphLaunch(_setup_graph_exec, _stream));
        gpuErrChk(hipStreamSynchronize(_stream));

        logCudaGraphNodes(_process_graph, spdlog::level::debug);
        gpuErrChk(hipGraphInstantiate(&_process_graph_exec, _process_graph, NULL, NULL, 0));
    }

    void process(const std::vector<float*>& dst_bufs, const std::vector<float*>& src_bufs) override {
        for (size_t i = 0; i < _input_vertices.size(); i++) {
            _input_vertices[i]->setSrcPtr({Buffer::create(src_bufs[i], BufferSpecs(MemoryContext::HOST, _n_proc_frames))}, _process_graph_exec);
        }
        for (size_t i = 0; i < _output_vertices.size(); i++) {
            _output_vertices[i]->setDestPtr({Buffer::create(dst_bufs[i], BufferSpecs(MemoryContext::HOST, _n_proc_frames))}, _process_graph_exec);
        }
        gpuErrChk(hipGraphLaunch(_process_graph_exec, _stream));
        gpuErrChk(hipStreamSynchronize(_stream));
    }
    void processAsync(const std::vector<float*>& dst_bufs, const std::vector<float*>& src_bufs) override {
        throw std::runtime_error("Not implemented");
    }

    void teardown() override {
        for (Buffer* buffer : _buffer_ptrs) {
            gpuErrChk(hipFree(buffer->getDataMod()));
        }
        gpuErrChk(hipStreamDestroy(_stream));
    }

    IGpuSignalVertex* add(IGpuFx* fx, IGpuSignalVertex* parent = nullptr) override {
        if (parent == nullptr && !_leaves.empty()) {
            parent = _leaves.front();
        }

        GpuSignalFxVertex* vertex = new GpuSignalFxVertex(fx, parent);
        if (parent != nullptr) {
            vertex->getChildren().insert(vertex->getChildren().end(), parent->getChildren().begin(), parent->getChildren().end());
            parent->getChildren().clear();
            parent->getChildren().push_back(vertex);
        }
        _vertices.push_back(vertex);
        updateLeaves();

        return vertex;
    }

    IGpuSignalVertex* addRoot(IGpuFx* fx) override {
        GpuSignalFxVertex* vertex = new GpuSignalFxVertex(fx, nullptr);
        _vertices.push_back(vertex);
        updateLeaves();

        return vertex;
    }

    std::vector<IGpuSignalVertex*> split(std::vector<IGpuFx*> fxs, IGpuSignalVertex* parent = nullptr) override {
        if (parent == nullptr && !_leaves.empty()) {
            parent = _leaves.front();
        }

        std::vector<IGpuSignalVertex*> vertices;
        for (auto fx : fxs) {
            GpuSignalFxVertex* vertex = new GpuSignalFxVertex(fx, parent);
            if (parent != nullptr) parent->getChildren().push_back(vertex);
            vertices.push_back(vertex);
            _vertices.push_back(vertex);
        }
        updateLeaves();
        return vertices;
    }

    IGpuSignalVertex* merge(IGpuFx* fx, std::vector<IGpuSignalVertex*> parents) override {
        GpuSignalFxVertex* vertex = new GpuSignalFxVertex(fx, parents);

        // std::vector<IGpuSignalVertex*> parents_with_children;
        // std::copy_if(parents.begin(), parents.end(), std::back_inserter(parents_with_children), [](IGpuSignalVertex* parent) { return !parent->getChildren().empty(); });
        // IGpuSignalVertex* parent_with_children = nullptr;
        // if (parents_with_children.size() > 1) {
        //     throw std::runtime_error("More than one parent has children. Cannot merge.");
        // } else if (parents_with_children.size() == 1) {
        //     parent_with_children = parents_with_children.front();
        //     vertex->getChildren().insert(vertex->getChildren().end(), parent_with_children->getChildren().begin(), parent_with_children->getChildren().end());
        // }

        for (auto parent : parents) {
            parent->getChildren().clear();
            parent->getChildren().push_back(vertex);
        }
        _vertices.push_back(vertex);

        updateLeaves();
        return vertex;
    }
};

IGpuSignalGraph* IGpuSignalGraph::createGpuSignalGraph() {
    return new GpuSignalGraph();
}
